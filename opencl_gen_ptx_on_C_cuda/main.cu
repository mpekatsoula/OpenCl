#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 10

int main(int argc,char *argv[]){

  if(argc<3){
    printf("Usage: ./test.cu <ptx_file> <cuda_device>\n");
    exit(0);
  }
  
  // Error code
  hipError_t error;

  int i;

  // Host variables
  float *h_A, *h_B, *h_C;
  h_A = (float *)malloc(sizeof(float)*SIZE);
  h_B = (float *)malloc(sizeof(float)*SIZE);
  h_C = (float *)malloc(sizeof(float)*SIZE);
  for (i=0;i<SIZE;i++){
    h_A[i] = 12;
    h_B[i] = i;
    h_C[i] = 0;
  }

  // Initialize driver API
  error = hipInit(0);
  if((int)error!=0){
    printf("Error! hipInit returned: %d\n",(int)error); 
    exit(0);
  }

  // Get Cuda Device and give handle
  hipDevice_t cu_device;
  error = hipDeviceGet(&cu_device,atoi(argv[2]));
  if((int)error!=0){
    printf("Error! hipDeviceGet returned: %d\n",(int)error);
    exit(0);
  }

  // Create context to run on device 
  hipCtx_t cu_context;
  error = hipCtxCreate(&cu_context, 0, cu_device);
  if((int)error!=0){
    printf("Error! hipCtxCreate returned: %d\n",(int)error);
    exit(0);
  }

  // Load ptx code
  hipModule_t cu_module;
  error = hipModuleLoad(&cu_module,argv[1]);
  if((int)error!=0){
    printf("Error! hipModuleLoad returned: %d\n",(int)error);
    exit(0);
  }

  // Get kernel function
  hipFunction_t func;
  error = hipModuleGetFunction(&func,cu_module,"VectorAdd"); 
  if((int)error!=0){
    printf("Error! hipModuleGetFunction returned: %d\n",(int)error);
    exit(0);
  }

  hipDeviceptr_t var1,var2,var3;

  // Allocate device memory
  unsigned int size = sizeof(float)*SIZE;

  error = hipMalloc(&var1, size);
  error = hipMalloc(&var2, size);
  error = hipMalloc(&var3, size);
  if((int)error!=0){
    printf("Error! hipMalloc returned: %d\n",(int)error);
    exit(0);
  }

  // Copy variables to host
  error = hipMemcpyHtoD(var1,h_C,size);
  error = hipMemcpyHtoD(var2,h_A,size);
  error = hipMemcpyHtoD(var3,h_B,size);
  if((int)error!=0){
    printf("Error! hipMemcpyHtoD returned: %d\n",(int)error);
    exit(0);
  }

  // Lauch kernel
  void *args[] = {&var1, &var2, &var3};
  error = hipModuleLaunchKernel(func, 10, 1, 1, 1, 1, 1, 0, NULL, args, NULL);
  if((int)error!=0){
    printf("Error! hipModuleLaunchKernel returned: %d\n",(int)error);
    exit(0);
  }

  // Get result to host
  error = hipMemcpyDtoH(h_C,var1,size);
  if((int)error!=0){
    printf("Error! hipMemcpyDtoH returned: %d\n",(int)error);
    exit(0);
  }

  // Free device memory
  error = hipFree(var1);
  error = hipFree(var2);
  error = hipFree(var3);
  if((int)error!=0){
    printf("Error! hipFree returned: %d\n",(int)error);
    exit(0);
  }

  // Destroy context
  error = hipCtxDestroy(cu_context);
  if((int)error!=0){
    printf("Error! hipCtxDestroy returned: %d\n",(int)error);
    exit(0);
  }


  // Print result
  for (i=0;i<SIZE;i++)
    printf("%f\n",h_C[i]);
}
