#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc,char *argv[]){

  if(argc<3){
    printf("Usage: ./test.cu <ptx_file> <cuda_device>\n");
    exit(0);
  }
  
  // Error code
  hipError_t error;
  
  // My number
  unsigned int h_var=7;
  
  // Initialize driver API
  error = hipInit(0);
  if((int)error!=0){
    printf("Error! hipInit returned: %d\n",(int)error); 
    exit(0);
  }

  // Get Cuda Device and give handle
  hipDevice_t cu_device;
  error = hipDeviceGet(&cu_device,atoi(argv[2]));
  if((int)error!=0){
    printf("Error! hipDeviceGet returned: %d\n",(int)error);
    exit(0);
  }

  // Create context to run on device 
  hipCtx_t cu_context;
  error = hipCtxCreate(&cu_context, 0, cu_device);
  if((int)error!=0){
    printf("Error! hipCtxCreate returned: %d\n",(int)error);
    exit(0);
  }

  // Load ptx code
  hipModule_t cu_module;
  error = hipModuleLoad(&cu_module,argv[1]);
  if((int)error!=0){
    printf("Error! hipModuleLoad returned: %d\n",(int)error);
    exit(0);
  }

  // Get kernel function
  hipFunction_t func;
  error = hipModuleGetFunction(&func,cu_module,"testing"); 
  if((int)error!=0){
    printf("Error! hipModuleGetFunction returned: %d\n",(int)error);
    exit(0);
  }

  hipDeviceptr_t var;

  // Allocate device memory
  unsigned int size = sizeof(unsigned int);
  error = hipMalloc(&var, size);
  if((int)error!=0){
    printf("Error! hipMalloc returned: %d\n",(int)error);
    exit(0);
  }

  // Copy variable to host
  error = hipMemcpyHtoD(var,&h_var,size);
  if((int)error!=0){
    printf("Error! hipMemcpyHtoD returned: %d\n",(int)error);
    exit(0);
  }

  // Lauch kernel
  void *args[] = {&var};
  error = hipModuleLaunchKernel(func, 1, 1, 1, 1, 1, 1, 0, NULL, args, NULL);
  if((int)error!=0){
    printf("Error! hipModuleLaunchKernel returned: %d\n",(int)error);
    exit(0);
  }

  // Get result to host
  error = hipMemcpyDtoH(&h_var,var,size);
  if((int)error!=0){
    printf("Error! hipMemcpyDtoH returned: %d\n",(int)error);
    exit(0);
  }

  // Free device memory
  error = hipFree(var);
  if((int)error!=0){
    printf("Error! hipFree returned: %d\n",(int)error);
    exit(0);
  }

  // Destroy context
  error = hipCtxDestroy(cu_context);
  if((int)error!=0){
    printf("Error! hipCtxDestroy returned: %d\n",(int)error);
    exit(0);
  }

  // Print result
  printf("var: %d\n",h_var);
}
